#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <sstream>
#include <assert.h>
#include <system_error>

// CUDA includes
#include <hip/hip_runtime.h>
#include <>

// cuBLAS includes
#include "hipblas.h"

/**************************************************/

// Function headers
float* cuda_lucaskanade(float* frame0, float* frame1, int w, int h);

// CUDA error handler
#define CUDA_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
	if (code != hipSuccess)
	{
		std::stringstream ss;
		ss << file << "(" << line << ")" << std::endl << hipGetErrorName(code) << " " << hipGetErrorString(code);
		
		std::string err_str;
		ss >> err_str;
		throw std::system_error(std::error_code(code, std::generic_category()), err_str.c_str());
	}
}

// MIPMAP GENERATION

uint32_t getMipMapLevels(hipExtent size) {
	uint32_t res = static_cast<uint32_t>(1 + std::floor(std::log2(std::max({ size.width, size.height, size.depth }))));
	return res;
}

__global__ void d_mipmap(hipSurfaceObject_t mipOutput, hipTextureObject_t mipInput, unsigned int imageW, unsigned int imageH) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float px = 1.0 / float(imageW);
	float py = 1.0 / float(imageH);


	if ((x < imageW) && (y < imageH))
	{
		// we are using the normalized access to make sure non-power-of-two textures
		// behave well when downsized.

		// Sample a 2x2 block next to current pixel
		float color =
			(tex2D<float>(mipInput, (x + 0) * px, (y + 0) * py)) +
			(tex2D<float>(mipInput, (x + 1) * px, (y + 0) * py)) +
			(tex2D<float>(mipInput, (x + 1) * px, (y + 1) * py)) +
			(tex2D<float>(mipInput, (x + 0) * px, (y + 1) * py));
		color /= 4.0;

		// Sample weighted 3x3 block centered at current pixel
		// This one seems to shift the image so I would avoid it
		//float color =
		//	0.25   * (tex2D<float>(mipInput, (x + 0) * px, (y + 0) * py)) +
		//	0.125  * (tex2D<float>(mipInput, (x + 1) * px, (y + 0) * py)) +
		//	0.125  * (tex2D<float>(mipInput, (x + 0) * px, (y + 1) * py)) +
		//	0.125  * (tex2D<float>(mipInput, (x + 0) * px, (y - 1) * py)) +
		//	0.125  * (tex2D<float>(mipInput, (x - 1) * px, (y + 0) * py)) +
		//	0.0625 * (tex2D<float>(mipInput, (x + 1) * px, (y + 1) * py)) +
		//	0.0625 * (tex2D<float>(mipInput, (x + 1) * px, (y - 1) * py)) +
		//	0.0625 * (tex2D<float>(mipInput, (x - 1) * px, (y + 1) * py)) +
		//	0.0625 * (tex2D<float>(mipInput, (x - 1) * px, (y - 1) * py));

		color = min(color, 1.0);

		surf2Dwrite(color, mipOutput, x * sizeof(float), y);
	}
}

void generateMipMaps(hipMipmappedArray_t mipmapArray, hipExtent extent) {
	size_t width = extent.width;
	size_t height = extent.height;

	unsigned int level = 0;

	while (width != 1 || height != 1)
	{
		width /= 2;
		width = std::max((size_t)1, width);
		height /= 2;
		height = std::max((size_t)1, height);

		hipArray_t levelFrom;
		CUDA_CALL(hipGetMipmappedArrayLevel(&levelFrom, mipmapArray, level));
		hipArray_t levelTo;
		CUDA_CALL(hipGetMipmappedArrayLevel(&levelTo, mipmapArray, level + 1));

		hipExtent  levelToSize;
		CUDA_CALL(hipArrayGetInfo(NULL, &levelToSize, NULL, levelTo));
		assert(levelToSize.width == width);
		assert(levelToSize.height == height);
		assert(levelToSize.depth == 0);

		// generate texture object for reading
		hipTextureObject_t         texInput;
		hipResourceDesc            texRes;
		memset(&texRes, 0, sizeof(hipResourceDesc));

		texRes.resType = hipResourceTypeArray;
		texRes.res.array.array = levelFrom;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));

		texDesc.normalizedCoords = 1;
		texDesc.filterMode = hipFilterModeLinear;

		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.addressMode[1] = hipAddressModeClamp;
		texDesc.addressMode[2] = hipAddressModeClamp;

		texDesc.readMode = hipReadModeElementType;

		CUDA_CALL(hipCreateTextureObject(&texInput, &texRes, &texDesc, NULL));

		// generate surface object for writing
		hipSurfaceObject_t surfOutput;
		hipResourceDesc    surfRes;
		memset(&surfRes, 0, sizeof(hipResourceDesc));
		surfRes.resType = hipResourceTypeArray;
		surfRes.res.array.array = levelTo;

		CUDA_CALL(hipCreateSurfaceObject(&surfOutput, &surfRes));

		// run mipmap kernel
		dim3 blockSize(16, 16, 1);
		dim3 gridSize(((unsigned int)width + blockSize.x - 1) / blockSize.x, ((unsigned int)height + blockSize.y - 1) / blockSize.y, 1);

		d_mipmap<<<gridSize, blockSize>>>(surfOutput, texInput, (unsigned int)width, (unsigned int)height);

		CUDA_CALL(hipDeviceSynchronize());
		CUDA_CALL(hipGetLastError());

		CUDA_CALL(hipDestroySurfaceObject(surfOutput));

		CUDA_CALL(hipDestroyTextureObject(texInput));

		level++;
	}
}

void setImageData(hipMipmappedArray_t mipmapArray, float* src, hipExtent extent) {
	size_t width = extent.width;
	size_t height = extent.height;

	// Upload level 0
	hipArray_t baseLevel;
	hipGetMipmappedArrayLevel(&baseLevel, mipmapArray, 0);
	hipMemcpyToArray(baseLevel, 0, 0, src, sizeof(float)*width*height, hipMemcpyHostToDevice);

	// Compute remaining mipmap levels
	generateMipMaps(mipmapArray, extent);
}

/***************************************************/

__global__ void lkKernel(float* result, hipTextureObject_t frame0, hipTextureObject_t frame1, const int width, const int height, int level)
{
	int windowSize = 5;

	float px = 1.0 / float(width);
	float py = 1.0 / float(height);

	unsigned int x = (blockIdx.x * blockDim.x + threadIdx.x);
	unsigned int y = (blockIdx.y * blockDim.y + threadIdx.y);
	unsigned int idx = y*width + x;

	float x0 = x >> level;
	float y0 = y >> level;

	result[idx * 2 + 0] = tex2DLod<float>(frame0, (x0) * px, (y0) * py, level);
	result[idx * 2 + 1] = tex2DLod<float>(frame1, (x0) * px, (y0) * py, level);

	return;

	if (x > width - 1 || y > height - 1) return;

	float det, D;

	float sum_Ixx = 0.0f;
	float sum_Ixy = 0.0f;
	float sum_Iyy = 0.0f;
	float Ix, Iy, It; // Image gradients

	level = 0;

	// Calculate spatial gradient
	for (int yy = -windowSize; yy <= windowSize; yy++) {
		for (int xx = -windowSize; xx <= windowSize; xx++) {
			Ix = tex2DLod<float>(frame0, (x0 + xx + 1) * px, (y0 + yy + 0) * py, level) - tex2DLod<float>(frame0, (x0 + xx - 1) * px, (y0 + yy + 0) * py, level);
			Iy = tex2DLod<float>(frame0, (x0 + xx + 0) * px, (y0 + yy + 1) * py, level) - tex2DLod<float>(frame0, (x0 + xx + 0) * px, (y0 + yy - 1) * py, level);
			
			sum_Ixx += Ix*Ix;
			sum_Ixy += Ix*Iy;
			sum_Iyy += Iy*Iy;
		}
	}

	det = sum_Ixx*sum_Iyy - sum_Ixy*sum_Ixy;

	if (det < 0.00001f) return;

	D = 1 / det;

	// Iterations
	float Vx = result[idx * 2 + 0];
	float Vy = result[idx * 2 + 1];

	float x1 = x + Vx;
	float y1 = y + Vy;

	float I, J;

	float sum_Ixt;
	float sum_Iyt;

	for (int iter = 0; iter < 5; iter++) {
		if (x1 < 0 || x1 > width - 1 || y1 < 0 || y1 > height - 1) return;

		sum_Ixt = 0.0f;
		sum_Iyt = 0.0f;

		for (int yy = -windowSize; yy <= windowSize; yy++) {
			for (int xx = -windowSize; xx <= windowSize; xx++) {
				I = tex2DLod<float>(frame0, (x0 + xx) * px, (y0 + yy) * py, level);
				J = tex2DLod<float>(frame1, (x1 + xx) * px, (y1 + yy) * py, level);

				Ix = tex2DLod<float>(frame0, (x0 + xx + 1) * px, (y0 + yy + 0) * py, level) - tex2DLod<float>(frame0, (x0 + xx - 1) * px, (y0 + yy + 0) * py, level);
				Iy = tex2DLod<float>(frame0, (x0 + xx + 0) * px, (y0 + yy + 1) * py, level) - tex2DLod<float>(frame0, (x0 + xx + 0) * px, (y0 + yy - 1) * py, level);

				It = J - I;

				sum_Ixt += Ix*It;
				sum_Iyt += Iy*It;
			}
		}

		float vx = D*(-sum_Iyy*sum_Ixt + sum_Ixy*sum_Iyt);
		float vy = D*( sum_Ixy*sum_Ixt - sum_Ixx*sum_Iyt);

		Vx += vx;
		Vy += vy;
		x1 += vx;
		y1 += vy;

		// Stop if movement is sufficiently small
		if (fabsf(vx) < 0.01f && fabsf(vy) < 0.01f) break;
	}

	if (level != 0) {
		Vx += Vx;
		Vy += Vy;
	}

	result[idx * 2 + 0] = I;// Vx;
	result[idx * 2 + 1] = J;// Vy;
}

float* cuda_lucaskanade(float* frame0, float* frame1, int w, int h) {
	float* result = new float[w*h*2];
	
	try {
		// Set CUDA device
		CUDA_CALL(hipSetDevice(0));

		// Allocate image buffers on GPU
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
		hipExtent imageExtent;
		imageExtent.width  = w;
		imageExtent.height = h;
		imageExtent.depth  = 0;

		hipMipmappedArray_t d_frame0;
		hipMipmappedArray_t d_frame1;

		CUDA_CALL(hipMallocMipmappedArray(&d_frame0, &channelDesc, imageExtent, getMipMapLevels(imageExtent)));
		CUDA_CALL(hipMallocMipmappedArray(&d_frame1, &channelDesc, imageExtent, getMipMapLevels(imageExtent)));

		// Set image data
		setImageData(d_frame0, frame0, imageExtent);
		setImageData(d_frame1, frame1, imageExtent);

		// Create the texture objects
		hipTextureObject_t d_tex0;
		hipTextureObject_t d_tex1;

		hipResourceDesc resDesc0;
		memset(&resDesc0, 0, sizeof(hipResourceDesc));

		resDesc0.resType = hipResourceTypeMipmappedArray;
		resDesc0.res.mipmap.mipmap = d_frame0;

		hipResourceDesc resDesc1;
		memset(&resDesc1, 0, sizeof(hipResourceDesc));

		resDesc1.resType = hipResourceTypeMipmappedArray;
		resDesc1.res.mipmap.mipmap = d_frame1;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));

		texDesc.normalizedCoords = 1;
		texDesc.filterMode = hipFilterModeLinear;
		texDesc.mipmapFilterMode = hipFilterModeLinear;

		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.addressMode[1] = hipAddressModeClamp;
		texDesc.addressMode[2] = hipAddressModeClamp;

		texDesc.maxMipmapLevelClamp = float(getMipMapLevels(imageExtent) - 1);

		texDesc.readMode = hipReadModeElementType;

		CUDA_CALL(hipCreateTextureObject(&d_tex0, &resDesc0, &texDesc, NULL));
		CUDA_CALL(hipCreateTextureObject(&d_tex1, &resDesc1, &texDesc, NULL));

		// Create intermediate resources
		float* d_dx;
		float* d_dy;
		CUDA_CALL(hipMalloc(&d_dx, sizeof(float)*w*h));
		CUDA_CALL(hipMalloc(&d_dy, sizeof(float)*w*h));

		CUDA_CALL(hipMemset(d_dx, 0, sizeof(float)*w*h));
		CUDA_CALL(hipMemset(d_dy, 0, sizeof(float)*w*h));

		// Create result resources
		float* d_result;
		CUDA_CALL(hipMalloc(&d_result, sizeof(float)*w*h*2));

		// Launch the kernel
		dim3 blockSize(16, 16, 1);
		dim3 gridSize(((unsigned int)w + blockSize.x - 1) / blockSize.x, ((unsigned int)h + blockSize.y - 1) / blockSize.y, 1);
		//for (int l = getMipMapLevels(imageExtent) - 1; l >= 0; l--) {
		//	lkKernel<<<gridSize, blockSize>>>(d_result, d_tex0, d_tex1, std::max(w>>l,1), std::max(h>>l,1), l);
		//	hipDeviceSynchronize();
		//}
		int l = 0;
		lkKernel << <gridSize, blockSize >> >(d_result, d_tex0, d_tex1, std::max(w >> l, 1), std::max(h >> l, 1), l);
		hipDeviceSynchronize();
		CUDA_CALL(hipPeekAtLastError());

		// Wait for kernel to finish
		CUDA_CALL(hipDeviceSynchronize());
		 
		// Retrieve output image from GPU
		CUDA_CALL(hipMemcpy(result, d_result, sizeof(float)*w*h*2, hipMemcpyDeviceToHost));

		// Free the buffers
		//CUDA_CALL(hipFree(d_frame0));
		//CUDA_CALL(hipFree(d_frame1));
		//CUDA_CALL(hipFree(d_result));
		//system("pause");
	}
	catch (std::system_error& e) {
		std::cerr << "CUDA ERROR " << e.code().value() << ": " << e.what() << std::endl;
		system("pause");
	}

	return result;
}